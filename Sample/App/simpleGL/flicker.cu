#include "hip/hip_runtime.h"
/* Flicker: An Interactive Particle Simulation Game using CUDA OpenGL Interop
 * 
 * Authored by Jayden Nyamiaka
 * 
 * Used Nvidia Sample's Simple GL as Base Structure 
 * 
 * 
 * Description
 * /////////////////////////////////////////////////////////////////////////////
 * This code runs a GPU-acceleratable particle simulation game that 
 * seamlessly integrates CUDA and OpenGL to get user-input, compute 
 * particle evolutions, and display results to the screen in real 
 * time. The behavior of particles is dependent on the user and needs
 * to be computed and rendered every frame, introducing significant 
 * challenges for continuously passing data between the CPU and GPU. 
 * 
 * The application currently supports 1 player and 3 different particle types.
 * Player: 
 *  - Controlled via WASD or Arrow Keys (simultaneously)
 *  - Color smoothly transitions over RGB spectrum
 *  - Can die on collision with particles. This should be on when using as a game.
 * 
 * 
 * Particles
 * Seeker (Particle 1):
 * This particle aims at the Player's position (at spawn time) and darts straight 
 * there with increasing acceleration. This particle changes colors from red to 
 * yellow depending on how fast it's traveling.
 * 
 * Cruiser (Particle 2):
 * This particle shoots in a simple (horizontal or vertical) direction but turns 
 * at seemingly random times. The turning is psuedo-random and frame-independent
 * implemented via a custom procedure that doesn't directly use any RNG. For 
 * implementation details, reference the code (and comments). This particle also 
 * changes colors depending on the direction its moving such that horizontally 
 * moving particles are green and vertically moving particles are violet.
 * 
 * Wanderer (Particle 3):
 * This particle wanders around by simulating Brownian motion, creating hot 
 * spots of dangerous unpredictability for the Player. Brownian motion models 
 * the random motion of particles suspended in a medium and is a mean zero, 
 * continuous process, often implemented using GPU acceleration due to its 
 * computational parallelism. This particle is also gray colored and slightly 
 * bigger than the others.

 *
 * For more information, refer to the code.
 * 
 * 
 * Usage
 * /////////////////////////////////////////////////////////////////////////////
 * The application doubles as both a game and a particle simulation, so the arguments we
 * opt to use for the application depend greatly on the use case. 
 * 
 * After building, the application can be called from the command line 
 * according to the following:
 * Usage: ./flicker.exe [options]
 * Options:
 * --help            Display this information.
 * --gpu-accel       Accelerate the simulation using the GPU. Otherwise, use the CPU.
 *                   Recommended for when using as a simulation.
 * --can-die         Has particle collisions kill the player & stop the simulation.
 *                   Otherwise, the player can't die. Recommended when using as a game.
 * --stagger         Stagger particle starts. Otherwise, all particles start simultaneously.
 *                   Recommended when using as a game.
 * --set-n n1 n2 n3  Manually set the number of each type of particle. Each n1 n2 n3 must
 *                   be a non-negative integer. The default is 32 32 32.
 * --preset [1-11]   Run the indicated simulation preset 1-11, discarding all other options.
 *                   The presets are as follows:
 *                     1:  Easy Game
 *                     2:  Hard Game
 *                     3:  Small Particle Simulation (All Particle Types)
 *                     4:  Large Particle Simulation (All Particle Types)
 *                     5:  Small Seeker Particle Simulation
 *                     6:  Large Seeker Particle Simulation
 *                     7:  Small Cruiser Particle Simulation
 *                     8:  Large Cruiser Particle Simulation
 *                     9:  Small Brownian Motion Simulation (Wanderer)
 *                     10: Large Brownian Motion Simulation (Wanderer)
 *                     11: Huge Seeker Particle Simulation
 * 
 * 
 * The presets list a few recommended calls depending on how you would like to use 
 * the application. Of course, you're encouraged to experiment with arguments and 
 * change the behavior of the application any way you like.
 * 
 * For all of the above presets, gpu_accel is toggled on depending on whether 
 * the application can benefit from GPU usage. In general, larger simulations 
 * benefitted from GPU acceleration due to the large number of particles that 
 * could all be evolved in parallel whereas the game and smaller simulations 
 * had too few particles to see any significant time improvements.
*/

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <map>
#include <math.h>
#include <random>

#define _USE_MATH_DEFINES


#ifdef _WIN32
#  define WINDOWS_LEAN_AND_MEAN
#  define NOMINMAX
#  include <windows.h>
#endif

// OpenGL Graphics includes
#include <helper_gl.h>
#if defined (__APPLE__) || defined(MACOSX)
  #pragma clang diagnostic ignored "-Wdeprecated-declarations"
  #include <GLUT/glut.h>
  #ifndef glutCloseFunc
  #define glutCloseFunc glutWMCloseFunc
  #endif
#else
#include <GL/freeglut.h>
#endif

// includes, cuda
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include <hiprand/hiprand_kernel.h>      // includes cuRand, cuda random support

// Utilities and timing functions
#include <helper_functions.h>    // includes hip/hip_runtime.h and hip/hip_runtime_api.h

// CUDA helper functions
#include <hip/hip_runtime_api.h>         // helper functions for CUDA error check

#include <hip/hip_vector_types.h>

#define MAX_EPSILON_ERROR 10.0f
#define THRESHOLD          0.30f
#define REFRESH_DELAY     10 //ms

#define SQRT2DIV2 0.70710678118f // Used as diagonal speed factor

#define MAX(a,b) ((a > b) ? a : b)


// Toggle GPU Acceleration On and Off (Can also be changed from Cmd Line)
////////////////////////////////////////////////////////////////////////////////

////////////////////////////////////////////////////////////////////////////////
//! Command Line Arguments (Currently set to default)
////////////////////////////////////////////////////////////////////////////////

// TODO Fix computeFPS
// TODO: R0Un Test to show time performance

// Count for each type of particle and total
// Defaults to the following values if not specified on cmd line
bool gpu_accel = false;
bool player_can_die = false;
bool particle_stagger_start = false;
unsigned int n_seekers =    32;     // Particle 1
unsigned int n_cruisers =   32;     // Particle 2
unsigned int n_wanderers =  32;     // Particle 3 1000000
unsigned int n_particles = n_seekers + n_cruisers + n_wanderers;


////////////////////////////////////////////////////////////////////////////////
//! Command Line Argument Functions
////////////////////////////////////////////////////////////////////////////////
void printUsage(char *exeName) {
    printf("Usage: %s [options]\n", exeName);
    printf("Options:\n");
    printf("--help            Display this information.\n");
    printf("--gpu-accel       Accelerate the simulation using the GPU. Otherwise, use the CPU. \n");
    printf("                  Recommended for when using as a simulation.\n");
    printf("--can-die         Has particle collisions kill the player & stop the simulation. \n");
    printf("                  Otherwise, the player can't die. Recommended when using as a game.\n");
    printf("--stagger         Stagger particle starts. Otherwise, all particles start simultaneously.\n");
    printf("                  Recommended when using as a game.\n");
    printf("--set-n n1 n2 n3  Manually set the number of each type of particle. Each n1 n2 n3 must \n");
    printf("                  be a non-negative integer. The default is %u %u %u.\n", n_seekers, n_cruisers, n_wanderers);
    printf("--preset [1-11]   Run the indicated simulation preset 1-11, discarding all other options. \n");
    printf("                  The presets are as follows: \n");
    printf("                    1:  Easy Game \n");
    printf("                    2:  Hard Game \n");
    printf("                    3:  Small All Particle Simulation \n");
    printf("                    4:  Large All Particle Simulation \n");
    printf("                    5:  Small Seeker Particle Simulation \n");
    printf("                    6:  Large Seeker Particle Simulation \n");
    printf("                    7:  Small Cruiser Particle Simulation \n");
    printf("                    8:  Large Cruiser Particle Simulation \n");
    printf("                    9:  Small Brownian Motion Simulation (Wanderer) \n");
    printf("                    10: Large Brownian Motion Simulation (Wanderer) \n");
    printf("                    11: Huge Seeker Particle Simulation \n");
}
void setSimulationConfig(bool gpu, bool can_die, bool stagger, int ns, int nc, int nw) {
    gpu_accel = gpu;
    player_can_die = can_die;
    particle_stagger_start = stagger;
    n_seekers =    ns;
    n_cruisers =   nc;
    n_wanderers =  nw;
    n_particles = n_seekers + n_cruisers + n_wanderers;
}
void setSimulationPreset(int i, char *execName) {
    switch (i) {
        case 1: // Easy Game
            setSimulationConfig(false, true, true, 16, 16, 16);
            break;
        case 2: // Hard Game
            setSimulationConfig(false, true, true, 32, 32, 32);
            break;
        case 3: // Small All Particle Simulation
            setSimulationConfig(false, false, false, 256, 256, 256);
            break;
        case 4: // Large All Particle Simulation
            setSimulationConfig(true, false, false, 65536, 65536, 65536);
            break;
        case 5: // Small Seeker Particle Simulation
            setSimulationConfig(false, false, false, 1000, 0, 0);
            break;
        case 6: // Large Seeker Particle Simulation
            setSimulationConfig(true, false, false, 100000, 0, 0);
            break;
        case 7: // Small Cruiser Particle Simulation
            setSimulationConfig(false, false, false, 0, 1000, 0);
            break;
        case 8: // Large Cruiser Particle Simulation
            setSimulationConfig(true, false, false, 0, 100000, 0);
            break;
        case 9: // Small Brownian Motion Simulation
            setSimulationConfig(false, false, false, 0, 0, 1000);
            break;
        case 10: // Large Brownian Motion Simulation
            setSimulationConfig(true, false, false, 0, 0, 100000);
            break;
        case 11: // Huge Seeker Particle Simulation
            setSimulationConfig(true, false, false, 5000000, 0, 0);
            break;
        default:
            std::cerr << "Error: --preset requires a positive integer argument from 1 to 13.\n";
            printUsage(execName);
            exit(1);
    }
}
void processCmdLineArgs(int argc, char *argv[]) {
    for (int i = 1; i < argc; ++i) {
        if (strcmp(argv[i], "--help") == 0) {
            printUsage(argv[0]);
            exit(0);
        } else if (strcmp(argv[i], "--gpu-accel") == 0) {
            gpu_accel = true;
        } else if (strcmp(argv[i], "--stagger") == 0) {
            particle_stagger_start = true;
        } else if (strcmp(argv[i], "--can-die") == 0) {
            player_can_die = true;
        } else if (strcmp(argv[i], "--set-n") == 0) {
            if (i + 3 < argc) {
                n_seekers = std::atoi(argv[i + 1]);
                n_cruisers = std::atoi(argv[i + 2]);
                n_wanderers = std::atoi(argv[i + 3]);
                n_particles = n_seekers + n_cruisers + n_wanderers;
                i += 3; // Skip next three arguments as they are processed
            } else {
                std::cerr << "Error: --set-n requires three non-negative integer arguments.\n";
                printUsage(argv[0]);
                exit(1);
            }
        } else if (strcmp(argv[i], "--preset") == 0) {
            if (i + 1 < argc) {
                setSimulationPreset(std::atoi(argv[i + 1]), argv[0]);
                i++; // Skip next argument as it is processed
            } else {
                std::cerr << "Error: --preset requires a positive integer argument from 1 to 11.\n";
                printUsage(argv[0]);
                exit(1);
            }
        } else {
            std::cerr << "Error: Unknown option " << argv[i] << "\n";
            printUsage(argv[0]);
            exit(1);
        }
    }
}

// SCREEN AND GENERAL GAME CONSTANTS
////////////////////////////////////////////////////////////////////////////////
const char *SIMULATION_NAME = "Flicker";

// Window Dimensions are the same as Viewport Dimensions
const unsigned int WINDOW_WIDTH  = 800;
const unsigned int WINDOW_HEIGHT = 800;
const float SCALEX = 2.0f / WINDOW_WIDTH;
const float SCALEY = 2.0f / WINDOW_HEIGHT;
// Amount of extra space to render off screen (in NDC)
const float EXCESS_RENDER = 0.2f;

// All coordinates and measurements are stored in NDC
// Multiply by SCALEX and SCALEY to convert from pixel space to NDC


// PLAYER CONSTANTS
////////////////////////////////////////////////////////////////////////////////
const float PLAYER_WIDTH = 16.f * SCALEX;
const float PLAYER_HEIGHT = 24.f * SCALEY;
const float PLAYER_SPEED = 0.7f;
const float PLAYER_COLOR_FACTOR = 3.0f;


// PARTICLE CONSTANTS
////////////////////////////////////////////////////////////////////////////////
// how many particles of each type spawn each second
const float PARTICLE_STAGGER_PER_SEC = 1.5; // 3 
const float PARTICLE_POINT_SIZE = 7.f * SCALEX; // Square side-length in NDC
const float STARTING_BASE_SPEED = 0.086789f;
const float PARTICLE_BASE_SPEED_ACCEL = 0.000812345f;

// Seeker Specific Constants
// Each second seeker speed increases by this proportion of its inital velocity
// Note: This factor is multiplied by the starting x and y velocities respectively and 
// then kept constant. Acceleration must be proportional to x and y components
// of velocity to maintain the same direction 
const float SEEKER_ACCEL_FACTOR = 0.4f; 
const float SEEKER_COLOR_DIV_SCALE = 4.0f;

// Cruiser Specific Constants
// how much faster it is than particle base speed
const float CRUISER_FASTER_SPEED_RATIO = 3.f;
// how much distance to travel per checking if we should turn
const float CRUISER_CHECK_TURN_DIST = SCALEX; // setting to SCALE_X makes it checl once each pixel
// mutliplied by varying_pos to determine which decimal magnitude decides if we turn
const unsigned int CRUISER_DIST_SCALE = 1000; // setting 1000 has us check the thousands place
// controls turning freq, turns every TURNING_FREQ_CONSTANT checks 
const unsigned int CRUISER_TURNING_FREQ_INV = 499; 

// Wanderer Specific Constants
const float WANDERER_NO_INITIALIZATION_X = 0.2f;
const float WANDERER_NO_INITIALIZATION_Y = 0.2f;
const float WANDERER_FASTER_SPEED_RATIO = 1.0f;
const float WANDERER_BIGGER_SIZE_RATIO = 1.5f;



////////////////////////////////////////////////////////////////////////////////
// GPU COPIES OF RELEVANT PARTICLE CONSTANTS
////////////////////////////////////////////////////////////////////////////////
// General Constants
__constant__ float d_EXCESS_RENDER;
__constant__ float d_PARTICLE_POINT_SIZE;
__constant__ float d_STARTING_BASE_SPEED;
__constant__ float d_PARTICLE_BASE_SPEED_ACCEL;

// Seeker Specific Constants
__constant__ float d_SEEKER_ACCEL_FACTOR;
__constant__ float d_SEEKER_COLOR_DIV_SCALE;

// Cruiser Specific Constants
__constant__ float d_CRUISER_FASTER_SPEED_RATIO;
__constant__ float d_CRUISER_CHECK_TURN_DIST;
__constant__ unsigned int d_CRUISER_DIST_SCALE;
__constant__ unsigned int d_CRUISER_TURNING_FREQ_INV;

// Wanderer Specific Constants
__constant__ float d_WANDERER_NO_INITIALIZATION_X;
__constant__ float d_WANDERER_NO_INITIALIZATION_Y;
__constant__ float d_WANDERER_FASTER_SPEED_RATIO;
__constant__ float d_WANDERER_BIGGER_SIZE_RATIO;



// GPU CONSTANTS
////////////////////////////////////////////////////////////////////////////////
const unsigned int BLOCK_SIZE = 256;
const unsigned long long RANDOM_BASE_SEED = 1234;


// GENERAL GAME VARIABLES
////////////////////////////////////////////////////////////////////////////////
// Boolean for if display has started (Turned on by pressing space bar)
bool simulation_on = false;

// Variables for time
float totalTime = 0.0f;
float dt = 0.0f;
// Variables for FPS 
float timeOneSec = 0.0f;
float currentFPS = 0.0f;

// Used for keyboard handlers
// Keeps track of which keys are currently being pressed
std::map<unsigned char, bool> keyStates;
std::map<int, bool> specialKeyStates;


// GPU VARIABLES
////////////////////////////////////////////////////////////////////////////////
const unsigned int NUM_STREAMS = 3; // for number of particle types
hipStream_t streams[NUM_STREAMS];
hiprandState *d_rand_states;
bool *d_player_death_occured;


// PARTICLE VARIABLES
////////////////////////////////////////////////////////////////////////////////
// VBO Variables
GLuint particleVBO;
struct hipGraphicsResource *cuda_vbo_resource;
void *d_vbo_buffer = NULL;


// Field Variables
float particle_base_speed = STARTING_BASE_SPEED;
std::default_random_engine generator; // used for Brownian Motion

// Auto-Verification and FPS Vars
unsigned int frameCount = 0;
unsigned int g_TotalErrors = 0;
bool g_bQAReadback = false;
// Timer is used to track FPS and time taken to calcute and render each frame,
// not for totalTime and dt (glutGet(GLUT_ELAPSED_TIME) is used for that)
StopWatchInterface *timer = NULL;



enum Direction { LEFT, RIGHT, DOWN, UP };


// PLAYER CLASS
////////////////////////////////////////////////////////////////////////////////
class Player {
private:
    // Fields
    ////////////////////////////////////////////////////////////////////////////
    float pos_x;
    float pos_y;

    float color_r;
    float color_g;
    float color_b;

    float vel_x;
    float vel_y;

    float width;
    float height;

    bool alive;

    // Private Methods
    ////////////////////////////////////////////////////////////////////////////
    // Checks which keys are currently being pressed to set velocity
    // Bounds players movement by restricting directional movement off-screen
    // Diagonal velocities are multiplied by sqrt(2)/2 for a consistent magnitude
    // Note: Top-level updateTime must be called first
    void updateVelocity() {
        vel_x = 0.0f;
        vel_y = 0.0f;

        bool up = keyStates['w'] || specialKeyStates[GLUT_KEY_UP];
        bool lt = keyStates['a'] || specialKeyStates[GLUT_KEY_LEFT];
        bool dn = keyStates['s'] || specialKeyStates[GLUT_KEY_DOWN];
        bool rt = keyStates['d'] || specialKeyStates[GLUT_KEY_RIGHT];

        if (!(up && dn)) {
            if (up) {
                if (pos_y + height/2.f < 1.0f)
                    vel_y = PLAYER_SPEED;
            } else if (dn) {
                if (pos_y - height/2.f > -1.0f)
                    vel_y = -PLAYER_SPEED;
            }
        }
        if (!(lt && rt)) {
            if (lt) {
                if (pos_x - width/2.f > -1.0f)
                    vel_x = -PLAYER_SPEED;
            } else if (rt) {
                if (pos_x + width/2.f < 1.0f)
                    vel_x = PLAYER_SPEED;
            }
        }

        if (vel_y != 0 && vel_x != 0) {
            vel_y *= SQRT2DIV2;
            vel_x *= SQRT2DIV2;
        }
    }

    // Basic position update based on velocity and dt
    // updateVelocity must be called first
    void updatePosition() {
        pos_x += vel_x * dt;
        pos_y += vel_y * dt;
    }

    // Updates color of the Player (smoothing over rainbow colors)
    // Note: Top-level updateTime must be called first
    void updateColor() {
        float pi_float = static_cast<float>(M_PI);
        color_r = (sinf(totalTime * PLAYER_COLOR_FACTOR) + 1.0f) / 2.0f;
        color_g = (sinf(totalTime * PLAYER_COLOR_FACTOR + 2.0f * pi_float / 3.0f) + 1.0f) / 2.0f;
        color_b = (sinf(totalTime * PLAYER_COLOR_FACTOR + 4.0f * pi_float / 3.0f) + 1.0f) / 2.0f;
    }

public:
    // Public Methods
    ////////////////////////////////////////////////////////////////////////////
    Player(float px, float py, float w, float h) {
        pos_x = px;
        pos_y = py;

        width = w;
        height = h;

        // Player starts alive
        alive = true;

        // Player has no initial velocity
        vel_x = 0.0f;
        vel_y = 0.0f;

        // Colors set through method based on total time
        updateColor();
    }      

    // Getters 
    float getPosX() {
        return pos_x;
    }
    float getPosY() {
        return pos_y;
    }
    float getWidth() {
        return width;
    }
    float getHeight() {
        return height;
    }
    bool isAlive() {
        return alive;
    }

    // Setters
    void kill() {
        if (player_can_die) {
            alive = false;

            // Effectively stops all behavior except player's color smoothing
            simulation_on = false;
        }
    }
    // Updates the vel, pos, and color of the Player (in that order)
    // Note: Top-level updateTime must be called first
    void update() {
        updateVelocity();
        updatePosition();
        updateColor();
    }
    // Updates only the color of the Player
    // Useful for rendering Player before starting simulation
    // Note: Top-level updateTime must be called first
    void updateOnlyColor() {
        updateColor();
    }


    // Make a different function for this so player doesn't need to import OpenGL
    void render() { 
        glBegin(GL_QUADS);
            glColor3f(color_r, color_g, color_b);
            glVertex2f(pos_x - width/ 2.f, pos_y - height / 2.f);  // Bottom left
            glVertex2f(pos_x + width/ 2.f, pos_y - height / 2.f);  // Bottom right
            glVertex2f(pos_x + width/ 2.f, pos_y + height / 2.f);  // Top right
            glVertex2f(pos_x - width/ 2.f, pos_y + height / 2.f);  // Top left
        glEnd();
    }
};


// PARTICLE STRUCTS
////////////////////////////////////////////////////////////////////////////////
/** Base Particle Struct 
 *  Has attributes of all Particles, all Particles inherit from this Struct 
 *  Conceptually, we opt to use Structs instead of Classes because we want to 
 *  implement the behavior-defining algorithms for all particles at once (and 
 *  in cuda), so there are no methods to declare.
 *
 *  This structure (and the optional attributes) makes it relatively easy to 
 *  define new particles without having to implement a class and virtual 
 *  methods (due to the reasons above).
 *  
 *  Currently, the Simulation supports 3 Particle Child Structs: 
 *       - Seeker
 *       - Cruiser
 *       - Wanderer
 *  These are described and implemented directly below.
 */
struct Particle {
    float pos_x;
    float pos_y;

    float color_r;
    float color_g;
    float color_b;

    float vel_x;
    float vel_y;

    float attr_a;
    float attr_b;
    // Add more additional attributes as necessary
};

/** Seeker (Particle 1)
 *  This particle aims at the Player's position (at spawn time) 
 *  and darts straight there with increasing acceleration.
 *  Additional attributes:
 *  attr_a : accel_x
 *  attr_b : accel_y
 *  Component acceleration = component vel * factor to maintain direction of motion
 *  This particle also changes colors from red to yellow depending on much how
 *  faster it is, computed as a ratio of current speed to starting base speed
 */ 
struct Seeker   : Particle { };

/** Cruiser (Particle 2)
 *  This particle shoots in a simple (horizontal or vertical) direction 
 *  but turns at seemingly random times.
 *  Additional attributes:
 *  attr_a : distance_traveled_since_last_check
 *  This particle also changes colors depending the direction it's going,
 *  Horizontally moving particles are Green, vertically moving particles are Violet
 *  Psuedo-Random Frame-Indepedent Turning:
 *    There are 2 challenges with designing this behavior:
 *      - Want random turning w/out needing to keep regenerating random numbers
 *      - Need it to be frame-independent so different frame rates don't result in different behavior 
 *    Procedure: 
 *      In update, each time distance_traveled_since_last_check exceeds CRUISER_CHECK_TURN_DIST:
 *          Turn if ((int)(varying_position * 1000) % TURNING_FREQ_CONSTANT == 0)
 *              If turning, also use varying_pos to determine direction
 */ 
struct Cruiser  : Particle { };

/** Wanderer (Particle 3)
 *  This particle wanders around by simulating Brownian motion, creating hot 
 *  spots of dangerous unpredictability for the Player.
 *  Brownian motion models the random motion of particles suspended in a medium 
 *  and is a mean zero, continuous process, often implemented using GPU 
 *  acceleration due to its computational parallelism.
 *  Additional attributes: None
 *  This particle is gray colored.
 */ 
struct Wanderer : Particle { };


// Players
////////////////////////////////////////////////////////////////////////////////
Player *p1;


// Declarations
////////////////////////////////////////////////////////////////////////////////
// Set up and clean up
bool runSimulation(int argc, char **argv);

// GL functionality
bool initGL(int *argc, char **argv);
void createParticleVBO(GLuint *vbo);
void deleteParticleVBO(GLuint *vbo);

// Declaring Callbacks and Handlers
void display();
void redisplay();
void cleanup();
void handleKeyPress(unsigned char key, int x, int y);
void handleKeyRelease(unsigned char key, int x, int y);
void handleSpecialKeyPress(int key, int x, int y);
void handleSpecialKeyRelease(int key, int x, int y);

// Display callback helper functions needed for display
void updateTime();
void computeFPS();

// Utility functions for generating a random numbers on the CPU
float randomFloatCPU(float min, float max); // range: [min, max)
int randomIntCPU(int min, int max); // range: [min, max]


////////////////////////////////////////////////////////////////////////////////
//! Particle Helper Functions Defined on both the CPU and GPU
////////////////////////////////////////////////////////////////////////////////
// Returns true if the particle has collided with the player
__host__ __device__ bool detectPlayerCollision(float px, float py, float psize,
        float target_x, float target_y, float target_w, float target_h) {
    /// Calculates the edges of the particle
    float halfLength = psize / 2.f;
    float left = px - halfLength;
    float right = px + halfLength;
    float top = py + halfLength;
    float bottom = py - halfLength;

    // Calculates the edges of the player
    float targetLeft = target_x - target_w / 2.0f;
    float targetRight = target_x + target_w / 2.0f;
    float targetTop = target_y + target_h / 2.0f;
    float targetBottom = target_y - target_h / 2.0f;

    // Checks for overlap
    return (left < targetRight && right > targetLeft
        && top > targetBottom && bottom < targetTop);
}

// Returns true if the particle is outside the screen bounds
// (including the excess render region)
__host__ __device__ bool detectOffScreen(float px, float py, 
        float psize, float excess_render) {
    // Calculates the edges of the particle
    float halfLength = psize / 2.f;
    float left = px - halfLength;
    float right = px + halfLength;
    float top = py + halfLength;
    float bottom = py - halfLength;

    float screen_min = -1.f - excess_render;
    float screen_max = 1.f + excess_render;

    // Checks if the particle is outside screen boundaries
    return (left < screen_min || right > screen_max
        || top > screen_max || bottom < screen_min);
}


////////////////////////////////////////////////////////////////////////////////
//! GPU Particle Functions
////////////////////////////////////////////////////////////////////////////////
int getNumBlocks(int n) {
    return (n + BLOCK_SIZE - 1) / BLOCK_SIZE;
}


//  GPU Set Up and Clean Up Functions
////////////////////////////////////////////////////////////////////////////////
void copyConstantsToGPU() {
    // Note: These don't need to be freed bc they are implicitly managed by CUDA runtime
    hipMemcpyToSymbol(HIP_SYMBOL(d_EXCESS_RENDER), &EXCESS_RENDER, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(d_PARTICLE_POINT_SIZE), &PARTICLE_POINT_SIZE, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(d_STARTING_BASE_SPEED), &STARTING_BASE_SPEED, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(d_PARTICLE_BASE_SPEED_ACCEL), &PARTICLE_BASE_SPEED_ACCEL, sizeof(float));

    hipMemcpyToSymbol(HIP_SYMBOL(d_SEEKER_ACCEL_FACTOR), &SEEKER_ACCEL_FACTOR, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(d_SEEKER_COLOR_DIV_SCALE), &SEEKER_COLOR_DIV_SCALE, sizeof(float));

    hipMemcpyToSymbol(HIP_SYMBOL(d_CRUISER_FASTER_SPEED_RATIO), &CRUISER_FASTER_SPEED_RATIO, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(d_CRUISER_CHECK_TURN_DIST), &CRUISER_CHECK_TURN_DIST, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(d_CRUISER_DIST_SCALE), &CRUISER_DIST_SCALE, sizeof(unsigned int));
    hipMemcpyToSymbol(HIP_SYMBOL(d_CRUISER_TURNING_FREQ_INV), &CRUISER_TURNING_FREQ_INV, sizeof(unsigned int));

    hipMemcpyToSymbol(HIP_SYMBOL(d_WANDERER_NO_INITIALIZATION_X), &WANDERER_NO_INITIALIZATION_X, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(d_WANDERER_NO_INITIALIZATION_Y), &WANDERER_NO_INITIALIZATION_Y, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(d_WANDERER_FASTER_SPEED_RATIO), &WANDERER_FASTER_SPEED_RATIO, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(d_WANDERER_BIGGER_SIZE_RATIO), &WANDERER_BIGGER_SIZE_RATIO, sizeof(float));
}
// CUDA kernel to initialize the random states
__global__ void init_rng_states(hiprandState *states, int n, int base_seed) {
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n) {
        hiprand_init(base_seed + id, id, 0, &states[id]);
    }
}
void setUpGPU(GLuint *p_vbo, struct hipGraphicsResource **vbo_res, 
        unsigned int vbo_res_flags) 
{
    // Copies all the constants on the host to the GPU constant memory
    copyConstantsToGPU();

    // Registers Particle VBO buffer object with CUDA
    checkCudaErrors(hipGraphicsGLRegisterBuffer(vbo_res, *p_vbo, vbo_res_flags));

    // Creates all streams
    for (int i = 0; i < NUM_STREAMS; i++) {
        checkCudaErrors(hipStreamCreate(&streams[i]));
    }

    // Initializes Random Number Generator for each particle
    checkCudaErrors( hipMalloc(&d_rand_states, n_particles * sizeof(hiprandState)) );
    init_rng_states<<<getNumBlocks(n_particles), BLOCK_SIZE>>>(
        d_rand_states, n_particles, RANDOM_BASE_SEED);

    // Initializes array to track player death in the GPU update function
    checkCudaErrors(hipMalloc((void**)&d_player_death_occured, 3 * sizeof(bool)));
}
void cleanUpGPU(struct hipGraphicsResource *vbo_res) {
    // Unregisters this buffer object with CUDA
    checkCudaErrors(hipGraphicsUnregisterResource(vbo_res));

    checkCudaErrors(hipFree(d_rand_states));
    checkCudaErrors(hipFree(d_player_death_occured));
}



//  GPU Random Functions
////////////////////////////////////////////////////////////////////////////////
// Device function to generate a random int between a and b (both inclusive)
__device__ int randomIntGPU(hiprandState *state, int a, int b) {
    float myrandf = hiprand_uniform(state);  // Uniform random float between 0 and 1
    return a + (int)((b - a + 1) * myrandf); // Scale and shift to [a, b]
}
// Device function to generate a random float between a and b (both inclusive)
__device__ float randomFloatGPU(hiprandState *state, float a, float b) {
    float myrandf = hiprand_uniform(state);  // Uniform random float between 0 and 1
    return a + (b - a) * myrandf;           // Scale and shift to [a, b]
}
// Device function to generate a random starting position and return the direction
__device__ Direction randomStartPosGPU(hiprandState *state, float* pos_x, float *pos_y) {
    Direction dir = static_cast<Direction>(randomIntGPU(state, 0, 3));

    switch (dir) {
        case LEFT: // Left edge
            *pos_x = -1.f - randomFloatGPU(state, d_PARTICLE_POINT_SIZE, d_EXCESS_RENDER);
            *pos_y = randomFloatGPU(state, -1.f, 1.f);
            break;
        case RIGHT: // Right edge
            *pos_x = 1.f + randomFloatGPU(state, d_PARTICLE_POINT_SIZE, d_EXCESS_RENDER);
            *pos_y = randomFloatGPU(state, -1.f, 1.f);
            break;
        case DOWN: // Bottom edge
            *pos_x = randomFloatGPU(state, -1.f, 1.f);
            *pos_y = -1.f - randomFloatGPU(state, d_PARTICLE_POINT_SIZE, d_EXCESS_RENDER);
            break;
        case UP: // Top edge
            *pos_x = randomFloatGPU(state, -1.f, 1.f);
            *pos_y = 1.f + randomFloatGPU(state, d_PARTICLE_POINT_SIZE, d_EXCESS_RENDER);
            break;
    }
    return dir;
}



//  GPU Seeker Functions
////////////////////////////////////////////////////////////////////////////////
__device__ void deviceInitSeeker(hiprandState *state, Seeker *s, 
        float particle_base_speed, float target_pos_x, float target_pos_y) 
{
    // Color (set to red, green changes in updateColor to get yellow)
    s->color_r = 1.0f;
    s->color_g = 0.0f;
    s->color_b = 0.0f;

    // Position
    randomStartPosGPU(state, &(s->pos_x), &(s->pos_y));

    // Velocity
    float dx = target_pos_x - s->pos_x;
    float dy = target_pos_y - s->pos_y;
    float dist = sqrtf(dx * dx + dy * dy);
    float ux = dx / dist;
    float uy = dy / dist;
    s->vel_x = ux * particle_base_speed;
    s->vel_y = uy * particle_base_speed;

    // Additional Attribute: Acceleration
    s->attr_a = s->vel_x * d_SEEKER_ACCEL_FACTOR;
    s->attr_b = s->vel_y * d_SEEKER_ACCEL_FACTOR;
}
__global__ void kernelInitSeekers(hiprandState *states, Seeker *seekers, int n, 
        float particle_base_speed, float target_pos_x, float target_pos_y) 
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= n) return;

    Seeker *s = &(seekers[idx]);
    hiprandState *localState = &(states[idx]);

    deviceInitSeeker(localState, s, particle_base_speed, target_pos_x, target_pos_y);
}
__global__ void kernelUpdateSeekers(hiprandState *states, Seeker *seekers, int n, 
        float particle_base_speed, bool *target_death, 
        float dt, bool staggerStart, int waitUntilBound,
        float target_x, float target_y, float target_w, float target_h) 
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= n) return;

    // Staggers starts when toggled on (for only Seeker and Cruiser)
    if (staggerStart && idx > waitUntilBound) return;

    Seeker *s = &(seekers[idx]);
    hiprandState *state = &(states[idx]);

    // Basic Velocity Update
    s->vel_x += s->attr_a * dt;
    s->vel_y += s->attr_b * dt;

    // Basic Position Update
    s->pos_x += s->vel_x * dt;
    s->pos_y += s->vel_y * dt;

    // Color Update: Hotter Effect
    // Increases the green value to make color go from red to yellow 
    // the faster it gets, simulating a "hotter", "more firey" look
    float speed = sqrtf(s->vel_x * s->vel_x + s->vel_y * s->vel_y);
    float ratioToStartingSpeed = speed / d_STARTING_BASE_SPEED - 1.f;
    s->color_g = (ratioToStartingSpeed - 1.f) / d_SEEKER_COLOR_DIV_SCALE;    

    // Do player collision first in case particle has collided with 
    // player a little outside of the screen bounds

    // Player Collision
    if (detectPlayerCollision(s->pos_x, s->pos_y, d_PARTICLE_POINT_SIZE,
            target_x, target_y, target_w, target_h)) {
        *target_death = true;
    }

    // If Offscreen -> Respawn
    if (detectOffScreen(s->pos_x, s->pos_y, d_PARTICLE_POINT_SIZE, d_EXCESS_RENDER)) {
        deviceInitSeeker(state, s, particle_base_speed, target_x, target_y);
    }
}



//  GPU Cruiser Functions
////////////////////////////////////////////////////////////////////////////////
__device__ void deviceInitCruiser(hiprandState *state, Cruiser *c, 
        float particle_base_speed) 
{
    Direction edge = randomStartPosGPU(state, &(c->pos_x), &(c->pos_y));

    // Velocity and Color
    c->vel_x = 0.0f;  
    c->vel_y = 0.0f;
    c->color_r = 0.0f;   // Sets horizontally moving to green
    c->color_g = 0.0f;   // Sets vertically moving to violet
    c->color_b = 0.0f;
    switch (edge) {
        case LEFT: // Left edge
            c->vel_x = particle_base_speed * d_CRUISER_FASTER_SPEED_RATIO;
            c->color_g = 1.0f;
            break;
        case RIGHT: // Right edge
            c->vel_x = -particle_base_speed * d_CRUISER_FASTER_SPEED_RATIO;
            c->color_g = 1.0f;
            break;
        case DOWN: // Bottom edge
            c->vel_y = particle_base_speed *  d_CRUISER_FASTER_SPEED_RATIO;
            c->color_r = 1.0f;
            c->color_b = 1.0f;
            break;
        case UP: // Top edge
            c->vel_y = -particle_base_speed * d_CRUISER_FASTER_SPEED_RATIO;
            c->color_r = 1.0f;
            c->color_b = 1.0f;
            break;
    }

    // Additional Attribute: Distance Traveled Since Last Check
    c->attr_a = 0.0f;
}
__global__ void kernelInitCruisers(hiprandState *states, Cruiser *cruisers, int n, 
        float particle_base_speed) 
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= n) return;

    Cruiser *c = &(cruisers[idx]);
    hiprandState *localState = &(states[idx]);

    deviceInitCruiser(localState, c, particle_base_speed);
}
__global__ void kernelUpdateCruisers(hiprandState *states, Cruiser *cruisers, int n, 
        float particle_base_speed, bool *target_death, 
        float dt, bool staggerStart, int waitUntilBound,
        float target_x, float target_y, float target_w, float target_h) 
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= n) return;

    // Staggers starts when toggled on (for only Seeker and Cruiser)
    if (staggerStart && idx > waitUntilBound) return;
    
    Cruiser *c = &(cruisers[idx]);
    hiprandState *state = &(states[idx]);

    // Position and Distance Traveled Since Last Check Update
    bool horizontal = (c->vel_x != 0.0f);
    if (horizontal) {
        float dx = c->vel_x * dt;
        c->pos_x += dx;
        c->attr_a += fabsf(dx);
    } else {
        float dy = c->vel_y * dt;
        c->pos_y += dy;
        c->attr_a += fabsf(dy);
    }

    // Psuedo-Random Frame-Indepedent Turning
    if (c->attr_a > d_CRUISER_CHECK_TURN_DIST) { // Then checks if we should turn
        c->attr_a = 0.0f; // Resets Distance Traveled Since Last Check
        float varying_pos = horizontal ? c->pos_x : c->pos_y;
        int turn_decider = (int)(varying_pos * d_CRUISER_DIST_SCALE);
        if (turn_decider % d_CRUISER_TURNING_FREQ_INV == 0) {
            float dir_change = ((turn_decider/10) % 2 == 0) ? -1.f : 1.f;
            if (horizontal) {
                // Speed Reversal (Horizontal to Vertical)
                c->vel_y = c->vel_x * dir_change;
                c->vel_x = 0.0f;
                // Color Change (Green to Violet)
                c->color_r = 1.0f;
                c->color_g = 0.0f;
                c->color_b = 1.0f;
            } else {
                // Speed Reversal (Vertical to Horizontal)
                c->vel_x = c->vel_y * dir_change;
                c->vel_y = 0.0f;
                // Color Change (Violet to Green)
                c->color_r = 0.0f;
                c->color_g = 1.0f;
                c->color_b = 0.0f;
            }
        }
    }

    // Do player collision first in case particle has collided with 
    // player a little outside of the screen bounds

    // Player Collision
    if (detectPlayerCollision(c->pos_x, c->pos_y, d_PARTICLE_POINT_SIZE,
            target_x, target_y, target_w, target_h)) {
        *target_death = true;
    }

    // If Offscreen -> Respawn
    if (detectOffScreen(c->pos_x, c->pos_y, d_PARTICLE_POINT_SIZE, d_EXCESS_RENDER)) {
        deviceInitCruiser(state, c, particle_base_speed);
    }
}



//  GPU Wanderer Functions
////////////////////////////////////////////////////////////////////////////////
__device__ void deviceInitWanderer(hiprandState *state, Wanderer *w, bool respawn,
        float particle_base_speed) 
{
    // Color (set to light gray)
    w->color_r = 0.8f;
    w->color_g = 0.8f;
    w->color_b = 0.8f;

    // Position
    if (respawn) {
        randomStartPosGPU(state, &(w->pos_x), &(w->pos_y));
    } else {
        w->pos_x = randomFloatGPU(state, d_WANDERER_NO_INITIALIZATION_X, 1.f + d_EXCESS_RENDER);
        w->pos_y = randomFloatGPU(state, d_WANDERER_NO_INITIALIZATION_Y, 1.f + d_EXCESS_RENDER);
        w->pos_x *= randomIntGPU(state, 0, 1) ? -1.f : 1.f;
        w->pos_y *= randomIntGPU(state, 0, 1) ? -1.f : 1.f;
    }

    // Velocity
    w->vel_x = particle_base_speed * d_WANDERER_FASTER_SPEED_RATIO;
    w->vel_y = particle_base_speed * d_WANDERER_FASTER_SPEED_RATIO;

    // Additional Attribute: None
}
__global__ void kernelInitWanderers(hiprandState *states, Wanderer *wanderers, int n, 
        float particle_base_speed) 
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= n) return;

    Wanderer *w = &(wanderers[idx]);
    hiprandState *localState = &(states[idx]);

    deviceInitWanderer(localState, w, false, particle_base_speed);
}
__global__ void kernelUpdateWanderers(hiprandState *states, Wanderer *wanderers, int n, 
    float particle_base_speed, bool *target_death, float dt, 
    float target_x, float target_y, float target_w, float target_h) 
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= n) return;

    // Wanderers don't stagger starts bc they spawn on screen

    Wanderer *w = &(wanderers[idx]);
    hiprandState *state = &(states[idx]);

    // Position Update (Brownian Motion Implementation)
    w->pos_x += hiprand_normal(state) * sqrtf(dt) * w->vel_x;
    w->pos_y += hiprand_normal(state) * sqrtf(dt) * w->vel_y;

    // Do player collision first in case particle has collided with 
    // player a little outside of the screen bounds

    // Player Collision
    if (detectPlayerCollision(w->pos_x, w->pos_y, 
            d_PARTICLE_POINT_SIZE * d_WANDERER_BIGGER_SIZE_RATIO,
            target_x, target_y, target_w, target_h)) {
        *target_death = true;
    }

    // If Offscreen -> Respawn
    if (detectOffScreen(w->pos_x, w->pos_y, d_PARTICLE_POINT_SIZE, d_EXCESS_RENDER)) {
        deviceInitWanderer(state, w, true, particle_base_speed);
    }
}



//  Main GPU Particle Functions
////////////////////////////////////////////////////////////////////////////////
void initParticlesGPU(struct hipGraphicsResource **vbo_resource) {
    hipDeviceSynchronize();

    // Maps the Particle VBO for writing from CUDA
    Particle *dvbo;
    checkCudaErrors(hipGraphicsMapResources(1, vbo_resource, 0));
    size_t num_bytes;
    checkCudaErrors(hipGraphicsResourceGetMappedPointer((void **)&dvbo, &num_bytes,
                                                         *vbo_resource));

    // Initializes all particles in the VBO                                           
    Seeker *d_seekers = static_cast<Seeker*>(dvbo);
    kernelInitSeekers<<<getNumBlocks(n_seekers), BLOCK_SIZE, 0, streams[0]>>>(
        d_rand_states, d_seekers, n_seekers, 
        particle_base_speed, p1->getPosX(), p1->getPosY());

    Cruiser *d_cruisers = static_cast<Cruiser*>(dvbo+n_seekers);
    kernelInitCruisers<<<getNumBlocks(n_cruisers), BLOCK_SIZE, 0, streams[1]>>>(
        d_rand_states+n_seekers, d_cruisers, n_cruisers,
        particle_base_speed);

    Wanderer *d_wanderers = static_cast<Wanderer*>(dvbo+n_seekers+n_cruisers);
    kernelInitWanderers<<<getNumBlocks(n_wanderers), BLOCK_SIZE, 0, streams[2]>>>(
        d_rand_states+n_seekers+n_cruisers, d_wanderers, n_wanderers,
        particle_base_speed);

    // Unmaps the Particle VBO
    checkCudaErrors(hipGraphicsUnmapResources(1, vbo_resource, 0));
}
void updateParticlesGPU(struct hipGraphicsResource **vbo_resource) {
    hipDeviceSynchronize();

    // Maps the Particle VBO for writing from CUDA
    Particle *dvbo;
    checkCudaErrors(hipGraphicsMapResources(1, vbo_resource, 0));
    size_t num_bytes;
    checkCudaErrors(hipGraphicsResourceGetMappedPointer((void **)&dvbo, &num_bytes,
                                                         *vbo_resource));                                                        

    // Used to stagger starts if toggled on through particle_stagger_start
    int waitUntilBound = static_cast<int>(totalTime * PARTICLE_STAGGER_PER_SEC);

    /* Make a separate player death bool for each stream to prevent sequential access
     * It's okay for mutiple threads in the same stream to access an element in 
     * d_player_death_occured bc they would both be changing it to the same value
     * bring true, so there's no race condition.
     */
    checkCudaErrors(hipMemset(d_player_death_occured, 0, 3 * sizeof(bool)));


    // Updates all particles in the VBO 
    Seeker *d_seekers = static_cast<Seeker*>(dvbo);
    kernelUpdateSeekers<<<getNumBlocks(n_seekers), BLOCK_SIZE, 0, streams[0]>>>(
        d_rand_states, d_seekers, n_seekers, 
        particle_base_speed, d_player_death_occured, 
        dt, particle_stagger_start, waitUntilBound, 
        p1->getPosX(), p1->getPosY(), p1->getWidth(), p1->getHeight());
            
    Cruiser *d_cruisers = static_cast<Cruiser*>(dvbo+n_seekers);
    kernelUpdateCruisers<<<getNumBlocks(n_cruisers), BLOCK_SIZE, 0, streams[1]>>>(
        d_rand_states+n_seekers, d_cruisers, n_cruisers,
        particle_base_speed, d_player_death_occured + 1, 
        dt, particle_stagger_start, waitUntilBound, 
        p1->getPosX(), p1->getPosY(), p1->getWidth(), p1->getHeight());

    Wanderer *d_wanderers = static_cast<Wanderer*>(dvbo+n_seekers+n_cruisers);
    kernelUpdateWanderers<<<getNumBlocks(n_wanderers), BLOCK_SIZE, 0, streams[2]>>>(
        d_rand_states+n_seekers+n_cruisers, d_wanderers, n_wanderers,
        particle_base_speed, d_player_death_occured + 2, dt,
        p1->getPosX(), p1->getPosY(), p1->getWidth(), p1->getHeight());
    

    // Kills player if any of the particles (in any of the 3 streams) collided with the player
    hipDeviceSynchronize();
    bool player_death_occured[3];
    checkCudaErrors(hipMemcpy(player_death_occured, d_player_death_occured, 3 * sizeof(bool), 
        hipMemcpyDeviceToHost));
    if (player_death_occured[0] || player_death_occured[1] || player_death_occured[2]) {
        p1->kill();
    }

    // Unmaps the Particle VBO
    checkCudaErrors(hipGraphicsUnmapResources(1, vbo_resource, 0));
}




////////////////////////////////////////////////////////////////////////////////
//! CPU Particle Functions
////////////////////////////////////////////////////////////////////////////////
// Starts PARTICLE_STAGGER_PER_SEC of each particle per second if particle_stagger_start is on
bool shouldDelayStart(int particleIdx) {
    if (!particle_stagger_start) {
        return false;
    }
    int waitUntilBound = static_cast<int>(totalTime * PARTICLE_STAGGER_PER_SEC); 
    return (particleIdx > waitUntilBound);
}

Direction randStartPosCPU(float &pos_x, float &pos_y);

void initSeekerCPU(Seeker &s);
void updateSeekerCPU(Seeker &s);

void initCruiserCPU(Cruiser &c);
void updateCruiserCPU(Cruiser &c);

void initWandererCPU(Wanderer &w, bool respawn);
void updateWandererCPU(Wanderer &w, std::normal_distribution<float> distribution);


bool initParticlesCPU() {
    // Maps the Particle VBO in order to modify it directly
    glBindBuffer(GL_ARRAY_BUFFER, particleVBO);
    Particle* ptr = (Particle*)glMapBuffer(GL_ARRAY_BUFFER, GL_READ_WRITE);
    if (!ptr) {
        fprintf(stderr, "ERROR: Failed to Map Particle Buffer for CPU Initialization.");
        fflush(stderr);
        g_TotalErrors++;
        return false;
    }

    unsigned int i = 0;
    while (i < n_seekers) {
        initSeekerCPU( ((Seeker*)ptr)[i++] );
    }
    while (i < n_seekers + n_cruisers) {
        initCruiserCPU( ((Cruiser*)ptr)[i++] );
    }
    while (i < n_seekers + n_cruisers + n_wanderers) {
        initWandererCPU( ((Wanderer*)ptr)[i++], false );
    }

    glUnmapBuffer(GL_ARRAY_BUFFER); // Unmaps the buffer when done
    return true;   
}


bool updateParticlesCPU() {
    // Maps the Particle VBO in order to modify it directly
    glBindBuffer(GL_ARRAY_BUFFER, particleVBO);
    Particle* ptr = (Particle*)glMapBuffer(GL_ARRAY_BUFFER, GL_READ_WRITE);
    if (!ptr) {
        fprintf(stderr, "ERROR: Failed to Map Particle Buffer for CPU Update.");
        fflush(stderr);
        g_TotalErrors++;
        return false;
    }
    
    unsigned int i = 0;
    // Updates Seeker Particles
    while (i < n_seekers) {
        if (shouldDelayStart(i)) {
            i++;
            continue;
        }
        updateSeekerCPU( ((Seeker*)ptr)[i++] );
    }
    // Updates Cruiser Particles
    while (i < n_seekers + n_cruisers) {
        if (shouldDelayStart(i - n_seekers)) {
            i++;
            continue;
        }
        updateCruiserCPU( ((Cruiser*)ptr)[i++] );
    }
    // Updates Wanderer Particles
    // In the rare case the sqrt(dt) is too small, we skip the update
    bool skipUpdate = false;
    float stddev = sqrtf(dt);
    if (stddev <= 0) {
        skipUpdate = true;
    }
    if (!skipUpdate) {
        std::normal_distribution<float> distribution(0.0f, stddev);
        while (i < n_seekers + n_cruisers + n_wanderers) {
            // No updates delay for wanderers since they are already on screen
            updateWandererCPU( ((Wanderer*)ptr)[i++], distribution );
        }
    }

    glUnmapBuffer(GL_ARRAY_BUFFER); // Unmaps the buffer when done
    return true; 
}



//  CPU Seeker Functions
////////////////////////////////////////////////////////////////////////////////
void initSeekerCPU(Seeker &s) {
    // Color (set to red, green changes in updateColor to get yellow)
    s.color_r = 1.0f;
    s.color_g = 0.0f;
    s.color_b = 0.0f;

    // Position
    randStartPosCPU(s.pos_x, s.pos_y);

    // Velocity
    float dx = p1->getPosX() - s.pos_x;
    float dy = p1->getPosY() - s.pos_y;
    float dist = sqrtf(dx * dx + dy * dy);
    float ux = dx / dist;
    float uy = dy / dist;
    s.vel_x = ux * particle_base_speed;
    s.vel_y = uy * particle_base_speed;

    // Additional Attribute: Acceleration
    s.attr_a = s.vel_x * SEEKER_ACCEL_FACTOR;
    s.attr_b = s.vel_y * SEEKER_ACCEL_FACTOR;
}


void updateSeekerCPU(Seeker &s) {
    // Basic Velocity Update
    s.vel_x += s.attr_a * dt;
    s.vel_y += s.attr_b * dt;

    // Basic Position Update
    s.pos_x += s.vel_x * dt;
    s.pos_y += s.vel_y * dt;

    // Color Update: Hotter Effect
    // Increases the green value to make color go from red to yellow 
    // the faster it gets, simulating a "hotter", "more firey" look
    float speed = sqrtf(s.vel_x * s.vel_x + s.vel_y * s.vel_y);
    float ratioToStartingSpeed = speed / STARTING_BASE_SPEED - 1.f;
    s.color_g = (ratioToStartingSpeed - 1.f) / SEEKER_COLOR_DIV_SCALE;

    // Do player collision first in case particle has collided with 
    // player a little outside of the screen bounds

    // Player Collision
    if (detectPlayerCollision(s.pos_x, s.pos_y, PARTICLE_POINT_SIZE,
            p1->getPosX(), p1->getPosY(), p1->getWidth(), p1->getHeight())) {
        p1->kill();
    }

    // If Offscreen -> Respawn
    if (detectOffScreen(s.pos_x, s.pos_y, PARTICLE_POINT_SIZE, EXCESS_RENDER)) {
        initSeekerCPU(s);
    }
}



//  CPU Cruiser Functions
////////////////////////////////////////////////////////////////////////////////
void initCruiserCPU(Cruiser &c) {
    // Position
    Direction edge = randStartPosCPU(c.pos_x, c.pos_y);

    // Velocity and Color
    c.vel_x = 0.0f;  
    c.vel_y = 0.0f;
    c.color_r = 0.0f;   // Sets horizontally moving to green
    c.color_g = 0.0f;   // Sets vertically moving to violet
    c.color_b = 0.0f;
    switch (edge) {
        case LEFT: // Left edge
            c.vel_x = particle_base_speed * CRUISER_FASTER_SPEED_RATIO;
            c.color_g = 1.0f;
            break;
        case RIGHT: // Right edge
            c.vel_x = -particle_base_speed * CRUISER_FASTER_SPEED_RATIO;
            c.color_g = 1.0f;
            break;
        case DOWN: // Bottom edge
            c.vel_y = particle_base_speed *  CRUISER_FASTER_SPEED_RATIO;
            c.color_r = 1.0f;
            c.color_b = 1.0f;
            break;
        case UP: // Top edge
            c.vel_y = -particle_base_speed * CRUISER_FASTER_SPEED_RATIO;
            c.color_r = 1.0f;
            c.color_b = 1.0f;
            break;
    }

    // Additional Attribute: Distance Traveled Since Last Check
    c.attr_a = 0.0f;
}


void updateCruiserCPU(Cruiser &c) {
    // Position and Distance Traveled Since Last Check Update
    bool horizontal = (c.vel_x != 0.0f);
    if (horizontal) {
        float dx = c.vel_x * dt;
        c.pos_x += dx;
        c.attr_a += fabsf(dx);
    } else {
        float dy = c.vel_y * dt;
        c.pos_y += dy;
        c.attr_a += fabsf(dy);
    }

    // Psuedo-Random Frame-Indepedent Turning
    if (c.attr_a > CRUISER_CHECK_TURN_DIST) { // Then checks if we should turn
        c.attr_a = 0.0f; // Resets Distance Traveled Since Last Check
        float varying_pos = horizontal ? c.pos_x : c.pos_y;
        int turn_decider = (int)(varying_pos * CRUISER_DIST_SCALE);
        if (turn_decider % CRUISER_TURNING_FREQ_INV == 0) {
            float dir_change = ((turn_decider/10) % 2 == 0) ? -1.f : 1.f;
            if (horizontal) {
                // Speed Reversal (Horizontal to Vertical)
                c.vel_y = c.vel_x * dir_change;
                c.vel_x = 0.0f;
                // Color Change (Green to Violet)
                c.color_r = 1.0f;
                c.color_g = 0.0f;
                c.color_b = 1.0f;
            } else {
                // Speed Reversal (Vertical to Horizontal)
                c.vel_x = c.vel_y * dir_change;
                c.vel_y = 0.0f;
                // Color Change (Violet to Green)
                c.color_r = 0.0f;
                c.color_g = 1.0f;
                c.color_b = 0.0f;
            }
        }
    }
    
    // Do player collision first in case particle has collided with 
    // player a little outside of the screen bounds

    // Player Collision
    if (detectPlayerCollision(c.pos_x, c.pos_y, PARTICLE_POINT_SIZE,
            p1->getPosX(), p1->getPosY(), p1->getWidth(), p1->getHeight())) {
        p1->kill();
    }

    // If Offscreen -> Respawn
    if (detectOffScreen(c.pos_x, c.pos_y, PARTICLE_POINT_SIZE, EXCESS_RENDER)) {
        initCruiserCPU(c);
    }
}



//  CPU Wanderer Functions
////////////////////////////////////////////////////////////////////////////////
void initWandererCPU(Wanderer &w, bool respawn) {
    // Color (set to light gray)
    w.color_r = 0.8f;
    w.color_g = 0.8f;
    w.color_b = 0.8f;

    // Position
    if (respawn) {
        randStartPosCPU(w.pos_x, w.pos_y);
    } else {
        w.pos_x = randomFloatCPU(WANDERER_NO_INITIALIZATION_X, 1.f + EXCESS_RENDER);
        w.pos_y = randomFloatCPU(WANDERER_NO_INITIALIZATION_Y, 1.f + EXCESS_RENDER);
        w.pos_x *= randomIntCPU(0, 1) ? -1.f : 1.f;
        w.pos_y *= randomIntCPU(0, 1) ? -1.f : 1.f;
    }

    // Velocity
    w.vel_x = particle_base_speed * WANDERER_FASTER_SPEED_RATIO;
    w.vel_y = particle_base_speed * WANDERER_FASTER_SPEED_RATIO;

    // Additional Attribute: None
}


void updateWandererCPU(Wanderer &w, std::normal_distribution<float> distribution) {
    // Doesn't use dt bc it's encoded into the creation of the distribution

    // Position Update (Brownian Motion Implementation)
    w.pos_x += distribution(generator) * w.vel_x;
    w.pos_y += distribution(generator) * w.vel_y;

    // Do player collision first in case particle has collided with 
    // player a little outside of the screen bounds

    // Player Collision
    if (detectPlayerCollision(w.pos_x, w.pos_y, 
            PARTICLE_POINT_SIZE * WANDERER_BIGGER_SIZE_RATIO,
            p1->getPosX(), p1->getPosY(), p1->getWidth(), p1->getHeight())) {
        p1->kill();
    }

    // If Offscreen -> Respawn
    if (detectOffScreen(w.pos_x, w.pos_y, 
            PARTICLE_POINT_SIZE * WANDERER_BIGGER_SIZE_RATIO, EXCESS_RENDER)) {
        // Respawns wanderers on the edge of the map instead of somewhere in the middle
        initWandererCPU(w, true);
    }
}



//  CPU Random Utility Functions
////////////////////////////////////////////////////////////////////////////////

// Generates a random starting position (in NDC, not pixels) somewhere on the 
// boundary of the screen by generating 3 random numbers (1 int, 2 floats).
// Returns which edge it placed the point at as a Direction
// One coordinate within EXCESS_RENDER region and other is within screen bounds
// Note: position is given in NDC
Direction randStartPosCPU(float &pos_x, float &pos_y) {
    Direction dir = static_cast<Direction>(randomIntCPU(0, 3));

    switch (dir) {
        case LEFT: // Left edge
            pos_x = -1.f - randomFloatCPU(PARTICLE_POINT_SIZE,EXCESS_RENDER);
            pos_y = randomFloatCPU(-1.f, 1.f);
            break;
        case RIGHT: // Right edge
            pos_x = 1.f + randomFloatCPU(PARTICLE_POINT_SIZE, EXCESS_RENDER);
            pos_y = randomFloatCPU(-1.f, 1.f);
            break;
        case DOWN: // Bottom edge
            pos_x = randomFloatCPU(-1.f, 1.f);
            pos_y = -1.f - randomFloatCPU(PARTICLE_POINT_SIZE,EXCESS_RENDER);
            break;
        case UP: // Top edge
            pos_x = randomFloatCPU(-1.f, 1.f);
            pos_y = 1.f + randomFloatCPU(PARTICLE_POINT_SIZE,EXCESS_RENDER);
            break;
    }
    return dir;
}

////////////////////////////////////////////////////////////////////////////////
// Generates a random float between min (inclusive) and max (exclusive)
float randomFloatCPU(float min, float max) {
    // Creates a random device and uses it to seed the RNG
    std::random_device rd;
    std::mt19937 gen(rd());

    // Define the distribution range [min, max)
    std::uniform_real_distribution<float> dis(min, max);

    // Generate and return the random float
    return dis(gen);
}

// Generates a random int between min (inclusive) and max (inclusive)
int randomIntCPU(int min, int max) {
    // Creates a random device and uses it to seed the RNG
    std::random_device rd;
    std::mt19937 gen(rd());

    // Define the distribution range [min, max]
    std::uniform_int_distribution<> dis(min, max);

    // Generate and return a random number within the range
    return dis(gen);
}


// Display Analytics and Errors 
void printSimulationAnalytics() { 
    printf("%s %s completed, took %f seconds, returned %s\n", SIMULATION_NAME, "Simulation", 
        totalTime, (g_TotalErrors == 0) ? "OK." : "ERROR!");

    printf("The simulation took an average of %f milliseconds per update", 
            sdkGetAverageTimerValue(&timer) / 1000.f);
    if (gpu_accel) {
        printf(" on the GPU.\n");
    } else {
        printf(" on the CPU.\n");
    }
}


////////////////////////////////////////////////////////////////////////////////
//! Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv)
{
    // Basic Linux preprocessor directive for display
    #if defined(__linux__)
    setenv ("DISPLAY", ":0", 0);
    #endif

    // Process Command Line Arguments
    processCmdLineArgs(argc, argv);

    // Print that the Simulation has started
    {
        printf("%s %s", SIMULATION_NAME, "Simulation");
        if (gpu_accel) {
            printf(" on the GPU");
        } else {
            printf(" on the CPU");
        }
        printf(" starting...\n");
    }

    // Run Simulation
    runSimulation(argc, argv);

    // Print Helpful Analytics from Simulation
    printSimulationAnalytics();

    exit(g_TotalErrors == 0 ? EXIT_SUCCESS : EXIT_FAILURE);
}



////////////////////////////////////////////////////////////////////////////////
//! Run the Interactive Particle Simulation
////////////////////////////////////////////////////////////////////////////////
bool runSimulation(int argc, char **argv)
{
    // Creates the CUTIL timer
    sdkCreateTimer(&timer);

    // Initializes GL, returning false on failure
    if (!initGL(&argc, argv)) {
        return false;
    }

    // Registers callbacks
    glutDisplayFunc(display);
    glutIdleFunc(redisplay);
    glutKeyboardFunc(handleKeyPress);
    glutKeyboardUpFunc(handleKeyRelease);
    glutSpecialFunc(handleSpecialKeyPress);
    glutSpecialUpFunc(handleSpecialKeyRelease);
    
    // Registers exit callback
    #if defined (__APPLE__) || defined(MACOSX)
    atexit(cleanup);
    #else
    glutCloseFunc(cleanup);
    #endif

    // Initializes Player
    p1 = new Player(0.0f, 0.0f, PLAYER_WIDTH, PLAYER_HEIGHT);

    // Creates the VBO that will store all the raw Particle data
    createParticleVBO(&particleVBO);

    // Sets up for GPU Acceleration
    if (gpu_accel) {
        setUpGPU(&particleVBO, &cuda_vbo_resource, cudaGraphicsMapFlagsWriteDiscard);
    }

    // Times the initialization
    sdkStartTimer(&timer);
    // Initializes Particles
    if (gpu_accel) {
        initParticlesGPU(&cuda_vbo_resource);
    } else {
        generator.seed(static_cast<unsigned int>(time(0)));
        bool particlesInitialized = initParticlesCPU();
        if (!particlesInitialized) return false;
    }
    sdkStopTimer(&timer);
    printf("Particle Initialization took %f seconds", sdkGetAverageTimerValue(&timer) / 1000.f);
    if (gpu_accel) {
        printf(" on the GPU.\n");
    } else {
        printf(" on the CPU.\n");
    }
    // Wipes timer so the initialization isn't considered in FPS computation
    sdkResetTimer(&timer);

    // Enters rendering mainloop
    glutMainLoop();

    // Returns true for success at the end
    return true;
}



////////////////////////////////////////////////////////////////////////////////
//! GL Functionality
////////////////////////////////////////////////////////////////////////////////

//  Set Up GL
////////////////////////////////////////////////////////////////////////////////
bool initGL(int *argc, char **argv)
{
    // Initialize GLUT and Create the Window
    glutInit(argc, argv);
    glutInitDisplayMode(GLUT_RGB | GLUT_DOUBLE);
    glutInitWindowSize(WINDOW_WIDTH, WINDOW_HEIGHT);
    glutCreateWindow(SIMULATION_NAME);

    // Handlers set during runSimulation

    // Initialize necessary OpenGL extensions
    if (! isGLVersionSupported(2,0))
    {
        fprintf(stderr, "ERROR: Support for necessary OpenGL extensions missing.");
        fflush(stderr);
        g_TotalErrors++;
        return false;
    }

    // Initialize a Blank (Black) Screen
    glClearColor(0.0f, 0.0f, 0.0f, 1.0f);

    // Window Dimensions are the same as Viewport Dimensions
    glViewport(0, 0, WINDOW_WIDTH, WINDOW_HEIGHT);

    SDK_CHECK_ERROR_GL();

    return true;
}


//  Create Particle VBO
////////////////////////////////////////////////////////////////////////////////
void createParticleVBO(GLuint *p_vbo)
{
    assert(p_vbo);

    // create buffer object
    glGenBuffers(1, p_vbo);

    // Bind the buffer to the target GL_ARRAY_BUFFER
    // Now all ops on GL_ARRAY_BUFFER will affect vbo (until unbound)
    glBindBuffer(GL_ARRAY_BUFFER, *p_vbo);

    // Initialize data for vbo (sets everything to NULL)
    unsigned int size = n_particles * sizeof(Particle);
    glBufferData(GL_ARRAY_BUFFER, size, 0, GL_DYNAMIC_DRAW);

    // Unbind vbo from target GL_ARRAY_BUFFER (binding to 0 unbinds the target)
    glBindBuffer(GL_ARRAY_BUFFER, 0);

    SDK_CHECK_ERROR_GL();
}


//  Delete Particle VBO
////////////////////////////////////////////////////////////////////////////////
void deleteParticleVBO(GLuint *p_vbo)
{

    glBindBuffer(1, *p_vbo);
    glDeleteBuffers(1, p_vbo);

    *p_vbo = 0;
}



////////////////////////////////////////////////////////////////////////////////
//! Callbacks and Handlers
////////////////////////////////////////////////////////////////////////////////

//! Display Callback
////////////////////////////////////////////////////////////////////////////////
void display()
{
    sdkStartTimer(&timer);

    // Updates total and delta time (in seconds)
    {
        // Gets seconds that have elapsed since glutInit was called
        float newTotalTime = glutGet(GLUT_ELAPSED_TIME) / 1000.f;
        dt = newTotalTime - totalTime;
        totalTime = newTotalTime;
        timeOneSec += dt; // for tracking FPS
    }

    // Increases Particle Base Speed over time (simulates harder game)
    {
        // If particle staggering is on, doesn't start increasing speed until
        // all particles have spawned in (started being updated)
        int n_most_particles = MAX(MAX(n_seekers, n_cruisers), n_wanderers);
        if (!particle_stagger_start
                || n_most_particles > totalTime * PARTICLE_STAGGER_PER_SEC) {
            particle_base_speed += PARTICLE_BASE_SPEED_ACCEL * dt;
        }
    }

    if (simulation_on) {
        p1->update();

        if (gpu_accel) {
            updateParticlesGPU(&cuda_vbo_resource);
        } else {
            updateParticlesCPU();
        }
    } else {
        p1->updateOnlyColor();
    }
    
    glClear(GL_COLOR_BUFFER_BIT);
    glClearColor(0.0f, 0.0f, 0.0f, 1.0f);

    p1->render();

    // Particle Rendering
    {
        // Tells OpenGL we'll be rendering vertices using array instead of individually 
        // Enable vertex position and color arrays
        glEnableClientState(GL_VERTEX_ARRAY);
        glEnableClientState(GL_COLOR_ARRAY);

        // Binds the particle VBO as target for vertex arrays below
        glBindBuffer(GL_ARRAY_BUFFER, particleVBO);

        // Specifies pointers to the vertex arrays using the bound buffer object
        glVertexPointer(2, GL_FLOAT, sizeof(Particle), (void*)0);
        glColorPointer(3, GL_FLOAT, sizeof(Particle), (void*)(2 * sizeof(GLfloat)));

        // Draws the Seeker and Cruiser particles at the normal size
        glPointSize(PARTICLE_POINT_SIZE / SCALEX);
        glDrawArrays(GL_POINTS, 0, n_seekers + n_cruisers);

        // Draws the Wanderer particles at the bigger indicated size
        glPointSize(PARTICLE_POINT_SIZE * WANDERER_BIGGER_SIZE_RATIO / SCALEX);
        glDrawArrays(GL_POINTS, n_seekers + n_cruisers, n_wanderers);

        // Disables the vertex arrays
        glDisableClientState(GL_VERTEX_ARRAY);
        glDisableClientState(GL_COLOR_ARRAY);
    }
        
    // Modelview matrix transform not yet used in this version

    // Swaps buffers, effecitvely rendering immediately
    glutSwapBuffers();

    sdkStopTimer(&timer);

    // Calculates fps using timer and displays it in window title
    computeFPS();
}


//! Close Callback
////////////////////////////////////////////////////////////////////////////////
void cleanup()
{
    printSimulationAnalytics();

    sdkDeleteTimer(&timer);

    if (gpu_accel) {
        cleanUpGPU(cuda_vbo_resource);
    }

    if (particleVBO)
    {
        deleteParticleVBO(&particleVBO);
    }

    if (p1) {
        delete p1;
    }
}


//! Idle Handler
////////////////////////////////////////////////////////////////////////////////
// Ensures continous display by always calling the display function
void redisplay() {
    glutPostRedisplay();
}


//! Keyboard Events Handlers
////////////////////////////////////////////////////////////////////////////////
// These handlers use 2 maps to track which keys are currently being held down
// Need 4 bc GL has separate handlers for ASCII/Non-ASCII Keys and Pressing/Releasing
void handleKeyPress(unsigned char key, int x, int y) {
    keyStates[key] = true;  // Set the state of the key to pressed
    switch (key) {
        case (' ') :
            simulation_on = true;
            break;
        case (27) :
            #if defined(__APPLE__) || defined(MACOSX)
                exit(EXIT_SUCCESS);
            #else
                glutDestroyWindow(glutGetWindow());
                return;
            #endif
    }

}
void handleKeyRelease(unsigned char key, int x, int y) {
    keyStates[key] = false; // Set the state of the key to not pressed
}
void handleSpecialKeyPress(int key, int x, int y) {
    specialKeyStates[key] = true;  // Set the state of the key to pressed
}
void handleSpecialKeyRelease(int key, int x, int y) {
    specialKeyStates[key] = false; // Set the state of the key to not pressed
}


//! Display Helper Functions
////////////////////////////////////////////////////////////////////////////////
// Computes and displays in Window Title once the simulation has started
// If the simulation has started, tells user to "Press Space to Start"
void computeFPS()
{
    char title[256];
    if (simulation_on) {
        frameCount++;
        if (timeOneSec >= 1.f)
        {
            currentFPS = static_cast<float>(frameCount) / timeOneSec;
            frameCount = 0;
            timeOneSec = 0.0f;
        }
        sprintf(title, "%s: %4.1f fps (Max 100Hz)", SIMULATION_NAME, currentFPS);
    } else {
        sprintf(title, "%s: Press Space to Start", SIMULATION_NAME);
    }
    glutSetWindowTitle(title);
}




/* Not Yet In Use 
    ////////////////////////////////////////////////////////////////////////////////
    //! Mouse event handlers
    // Not yet used for this version
    ////////////////////////////////////////////////////////////////////////////////

    //int mouse_old_x, mouse_old_y;
    //int mouse_buttons = 0;
    //float rotate_x = 0.0, rotate_y = 0.0;
    //float translate_z = -3.0;

    void mouse(int button, int state, int x, int y)
    {
        if (state == GLUT_DOWN)
        {
            mouse_buttons |= 1<<button;
        }
        else if (state == GLUT_UP)
        {
            mouse_buttons = 0;
        }

        mouse_old_x = x;
        mouse_old_y = y;
    }

    void motion(int x, int y)
    {
        float dx, dy;
        dx = (float)(x - mouse_old_x);
        dy = (float)(y - mouse_old_y);

        if (mouse_buttons & 1)
        {
            rotate_x += dy * 0.2f;
            rotate_y += dx * 0.2f;
        }
        else if (mouse_buttons & 4)
        {
            translate_z += dy * 0.01f;
        }

        mouse_old_x = x;
        mouse_old_y = y;
    }

    // To be used in the display function for a later version
    void transformModelView() {
        // Modelview matrix doesn't need to be changed for this version
        //glMatrixMode(GL_MODELVIEW);
        //glLoadIdentity();
        //glTranslatef(0.0, 0.0, translate_z);
        //glRotatef(rotate_x, 1.0, 0.0, 0.0);
        //glRotatef(rotate_y, 0.0, 1.0, 0.0);
    }
*/